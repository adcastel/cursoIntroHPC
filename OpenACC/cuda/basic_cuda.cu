#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 16


/////////////////////////////////////////////
// Codigo que se ejecuta en la GPU
///////////////////////////////////////////////
__global__ void simple(int * A, int n){

int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < n){

     A[idx] = idx;
   } 
}

int main(int argc, char * argv[]){

    int i;
    ///////////////////////////////////////////////
    // Reservamos memoria en la CPU
    //////////////////////////////////////////////
    int * A = (int *)malloc(N*sizeof(int));
    int * A_gpu;
    int block_size, num_blocks;
    /////////////////////////////////////////////
    // Numero de threads por bloque
    ////////////////////////////////////////////
    block_size = 4;
    
    /////////////////////////////////////////////
    // Numero de bloques
    ////////////////////////////////////////////
    num_blocks = N/block_size;
    
    ///////////////////////////////////////////////
    // Reservamos memoria en la GPU
    //////////////////////////////////////////////
    hipMalloc((void **)&A_gpu,N*sizeof(int));

    ///////////////////////////////////////////////
    // Movemos datos de la CPU a la GPU
    //////////////////////////////////////////////
    hipMemcpy(A_gpu, A, N*sizeof(int), hipMemcpyHostToDevice);

    /////////////////////////////////////////////
    // Ejecutamos el codigo con num_blocks bloques
    // de threads de block_size threads cada uno
    ///////////////////////////////////////////////
    simple<<<num_blocks,block_size>>>(A_gpu, N);
    
    ///////////////////////////////////////////////
    // Movemos datos de la GPU a la CPU
    //////////////////////////////////////////////
    hipMemcpy(A, A_gpu, N*sizeof(int),hipMemcpyDeviceToHost);
    
    for(i = 0; i < N-1; i++){
       printf("%d, ",A[i]);
    }
       printf("%d\n",A[N-1]);


    free(A);
    hipFree(A_gpu);

    return 1;
}


